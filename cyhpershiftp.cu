#include <iostream>
#include <string>
#include <hip/hip_runtime.h>

using namespace std;

// Kernel running on the GPU
__global__ void encryptCaesar(char *d_input, char *d_output, int shift, int length) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Print debug message from the GPU
    if (idx == 0) {
        printf("Running encryption on the GPU...\n");
    }

    if (idx < length) {
        // Encrypt each character
        char c = d_input[idx];

        // Check if the character is an uppercase letter
        if (c >= 'A' && c <= 'Z') {
            d_output[idx] = (c - 'A' + shift) % 26 + 'A';
        }
        // Check if the character is a lowercase letter
        else if (c >= 'a' && c <= 'z') {
            d_output[idx] = (c - 'a' + shift) % 26 + 'a';
        }
        else {
            // Non-alphabetical characters remain the same
            d_output[idx] = c;
        }
    }
}

int main() {
    string input;
    int shift;

    // Ask user for input string and shift value
    cout << "Enter the string to encrypt: ";
    getline(cin, input);
    cout << "Enter the shift value for Caesar cipher (1-25): ";
    cin >> shift;

    // CPU message
    cout << "Running setup and memory allocation on the CPU...\n";

    int length = input.length();
    char *d_input, *d_output;

    // Allocate memory on the CPU (host)
    char *h_input = new char[length + 1];
    char *h_output = new char[length + 1];  // Include null terminator for safety

    // Copy the input string to the host memory
    strcpy(h_input, input.c_str());

    // Allocate memory on the GPU (device)
    hipMalloc((void **)&d_input, length * sizeof(char));
    hipMalloc((void **)&d_output, length * sizeof(char));

    // Copy data from CPU to GPU
    hipMemcpy(d_input, h_input, length * sizeof(char), hipMemcpyHostToDevice);

    // Define the block size and grid size
    int blockSize = 256;
    int gridSize = (length + blockSize - 1) / blockSize;

    // Launch the kernel to encrypt the string
    cout << "Launching kernel on the GPU...\n";
    encryptCaesar<<<gridSize, blockSize>>>(d_input, d_output, shift, length);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy the result back to host memory
    hipMemcpy(h_output, d_output, length * sizeof(char), hipMemcpyDeviceToHost);

    // Add null terminator to output string
    h_output[length] = '\0';

    // Print the encrypted string
    cout << "Encrypted string: " << h_output << endl;

    // CPU message
    cout << "Cleaning up GPU memory on the CPU...\n";

    // Free the allocated GPU memory
    hipFree(d_input);
    hipFree(d_output);

    // Free the allocated CPU memory
    delete[] h_input;
    delete[] h_output;

    return 0;
}
